#include "hip/hip_runtime.h"
/*Copyright (c) 2014, Edgar Solomonik, all rights reserved.*/

#include <complex>
#include <assert.h>
#include <stdio.h>
#include "int_timer.h"
#include <stdint.h>

#include "offload.h"
#include "../tensor/algstrct.h"
#include "../interface/timer.h"

#ifdef USE_CUDA
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include ""
#endif

namespace CTF_int{
  volatile static int64_t int64_t_max = INT64_MAX;
  #ifndef PROFILE
  #define TAU_PROFILE(NAME,ARG,USER)
  #define TAU_PROFILE_TIMER(ARG1, ARG2, ARG3, ARG4)
  #define TAU_PROFILER_CREATE(ARG1, ARG2, ARG3, ARG4)
  #define TAU_PROFILE_STOP(ARG)
  #define TAU_PROFILE_START(ARG)
  #define TAU_PROFILE_SET_NODE(ARG)
  #define TAU_PROFILE_SET_CONTEXT(ARG)
  #define TAU_FSTART(ARG)
  #define TAU_FSTOP(ARG)
  #endif
  
  #define ABORT                                   \
    do{                                           \
     assert(0); } while (0)
  
#ifdef USE_CUDA
  int initialized = 0;
  hipblasHandle_t cuhandle;
  
  void offload_init(){
    if (!initialized){
      int ndev=0;
      hipGetDeviceCount(&ndev);
      assert(ndev > 0);
      hipblasStatus_t status = hipblasCreate(&cuhandle);
      assert(status == HIPBLAS_STATUS_SUCCESS);
    }
    initialized = 1;
  }
  
  void offload_exit(){
    if (initialized){
      hipblasStatus_t status = hipblasDestroy(cuhandle);
      assert(status == HIPBLAS_STATUS_SUCCESS);
      initialized = 0;
    }
  }
  
  offload_ptr::offload_ptr(algstrct const * sr_, int64_t size_){
    sr = sr_;
    size = size_;
    hipError_t err = hipMalloc((void**)&dev_ptr, size*sr->el_size);
    assert(err == hipSuccess);
  }
  
  offload_ptr::~offload_ptr(){
    hipError_t err = hipFree(dev_ptr);
    assert(err == hipSuccess);
  }

  LinModel<2> upload_mdl(upload_mdl_init,"upload_mdl");
  LinModel<2> download_mdl(download_mdl_init,"download_mdl");

  double estimate_download_time(int64_t size){
    double ps[] = {1.0, (double)size};
    return download_mdl.est_time(ps);
  }

  double estimate_upload_time(int64_t size){
    double ps[] = {1.0, (double)size};
    return upload_mdl.est_time(ps);
  }
  
  void offload_ptr::download(char * host_ptr){
    assert(initialized);
    TAU_FSTART(cuda_download);
    double st_time = MPI_Wtime();
    hipError_t err = hipMemcpy(host_ptr, dev_ptr, size*sr->el_size,
                                 hipMemcpyDeviceToHost);
    double exe_time = MPI_Wtime()-st_time;
    double tps[] = {exe_time, 1.0, (double)size*sr->el_size};
    download_mdl.observe(tps);
    TAU_FSTOP(cuda_download);
    assert(err == hipSuccess);
  }
  
  void offload_ptr::upload(char const * host_ptr){
    TAU_FSTART(cuda_upload);
    double st_time = MPI_Wtime();
    hipError_t err = hipMemcpy(dev_ptr, host_ptr, size*sr->el_size,
                                 hipMemcpyHostToDevice);

    double exe_time = MPI_Wtime()-st_time;
    double tps[] = {exe_time, 1.0, (double)size*sr->el_size};
    upload_mdl.observe(tps);
    TAU_FSTOP(cuda_upload);
    assert(err == hipSuccess);
  }
  
  
  template <typename dtype>
  __global__ void gset_zero(dtype *arr, int64_t size, dtype val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
    for (int i=idx; i<size; i+= gridDim.x*blockDim.x) {
      arr[i]=val;
    }
  }
  
  void offload_ptr::set_zero(){
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / (size);
    switch (sr->el_size){
      case 4:
        gset_zero<<<blockSize, numBlocks>>>((float*)dev_ptr, size, ((float*)sr->addid())[0]);
        break;
      case 8:
        gset_zero<<<blockSize, numBlocks>>>((double*)dev_ptr, size, ((double*)sr->addid())[0]);
        break;
      case 16:
        gset_zero<<<blockSize, numBlocks>>>((std::complex<double>*)dev_ptr, size, ((std::complex<double>*)sr->addid())[0]);
        break;
      default:
        assert(0);
        break;
    }
  }
  
  void host_pinned_alloc(void ** ptr, int64_t size){
    hipError_t err = hipHostAlloc(ptr, size, hipHostMallocMapped);
    assert(err == hipSuccess);
  }
  
  void host_pinned_free(void * ptr){
    hipError_t err = hipHostFree(ptr);
    assert(err == hipSuccess);
  }
#endif
 

  template 
  void offload_gemm(char          tA,
                    char          tB,
                    int           m,
                    int           n,
                    int           k,
                    double        alpha,
                    offload_ptr & A,
                    int           lda_A,
                    offload_ptr & B,
                    int           lda_B,
                    double        beta,
                    offload_ptr & C,
                    int           lda_C);
  template 
  void offload_gemm(char                 tA,
                    char                 tB,
                    int                  m,
                    int                  n,
                    int                  k,
                    std::complex<double> alpha,
                    offload_ptr &        A,
                    int                  lda_A,
                    offload_ptr &        B,
                    int                  lda_B,
                    std::complex<double> beta,
                    offload_ptr &        C,
                    int                  lda_C);

  template <>
  void offload_gemm<double>(char           tA,
                            char           tB,
                            int            m,
                            int            n,
                            int            k,
                            double         alpha,
                            double const * dev_A,
                            int            lda_A,
                            double const * dev_B,
                            int            lda_B,
                            double         beta,
                            double *       dev_C,
                            int            lda_C){
  #ifdef USE_CUDA
    assert(initialized);
  
    hipblasOperation_t cuA;  
    switch (tA){
      case 'n':
      case 'N':
        cuA = HIPBLAS_OP_N;
        break;
      case 't':
      case 'T':
        cuA = HIPBLAS_OP_T;
        break;
    }  
  
    hipblasOperation_t cuB;
    switch (tB){
      case 'n':
      case 'N':
        cuB = HIPBLAS_OP_N;
        break;
      case 't':
      case 'T':
        cuB = HIPBLAS_OP_T;
        break;
    }  
  
    hipblasStatus_t status = 
      hipblasDgemm(cuhandle, cuA, cuB, m, n, k, &alpha, 
                  dev_A, lda_A, 
                  dev_B, lda_B, &beta, 
                  dev_C, lda_C);
  #ifdef PROFILE
    hipDeviceSynchronize();
  #endif
    
    assert(status == HIPBLAS_STATUS_SUCCESS);
  #endif  
  }
  
  
  template <>
  void offload_gemm< std::complex<double> >(
                           char                         tA,
                           char                         tB,
                           int                          m,
                           int                          n,
                           int                          k,
                           std::complex<double>         alpha,
                           std::complex<double> const * dev_A,
                           int                          lda_A,
                           std::complex<double> const * dev_B,
                           int                          lda_B,
                           std::complex<double>         beta,
                           std::complex<double> *       dev_C,
                           int                          lda_C){
  #ifdef USE_CUDA
    assert(initialized);
    
    hipblasOperation_t cuA;  
    switch (tA){
      case 'n':
      case 'N':
        cuA = HIPBLAS_OP_N;
        break;
      case 't':
      case 'T':
        cuA = HIPBLAS_OP_T;
        break;
      case 'c':
      case 'C':
        cuA = HIPBLAS_OP_C;
        break;
    }  
  
    hipblasOperation_t cuB;
    switch (tB){
      case 'n':
      case 'N':
        cuB = HIPBLAS_OP_N;
        break;
      case 't':
      case 'T':
        cuB = HIPBLAS_OP_T;
        break;
      case 'c':
      case 'C':
        cuB = HIPBLAS_OP_C;
        break;
    }  
  
    TAU_FSTART(cublas_zgemm);
    hipblasStatus_t status = 
      hipblasZgemm(cuhandle, cuA, cuB, m, n, k, 
                  reinterpret_cast<hipDoubleComplex*>(&alpha), 
                  reinterpret_cast<const hipDoubleComplex*>(dev_A), lda_A, 
                  reinterpret_cast<const hipDoubleComplex*>(dev_B), lda_B, 
                  reinterpret_cast<hipDoubleComplex*>(&beta), 
                  reinterpret_cast<hipDoubleComplex*>(dev_C), lda_C);
  #ifdef PROFILE
    hipDeviceSynchronize();
  #endif
    TAU_FSTOP(cublas_zgemm);
    
    assert(status == HIPBLAS_STATUS_SUCCESS);
    assert(status == HIPBLAS_STATUS_SUCCESS);
  #endif
  }
 
  template <typename dtype>
  void offload_gemm(char           tA,
                    char           tB,
                    int            m,
                    int            n,
                    int            k,
                    dtype          alpha,
                    offload_ptr &  A,
                    int            lda_A,
                    offload_ptr &  B,
                    int            lda_B,
                    dtype          beta,
                    offload_ptr &  C,
                    int            lda_C){
    TAU_FSTART(cuda_gemm);
    offload_gemm(tA, tB, m, n, k, alpha, (dtype*)A.dev_ptr, lda_A, (dtype*)B.dev_ptr, lda_B, beta, (dtype*)C.dev_ptr, lda_C);
    TAU_FSTOP(cuda_gemm);
  }
}
